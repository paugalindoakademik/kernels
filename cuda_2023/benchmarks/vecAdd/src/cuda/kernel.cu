
#include <hip/hip_runtime.h>
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 *cr   This version maintained by: Nasser Anssari (anssari1@illinois.edu)
 ***************************************************************************/

__global__ void vecAdd(float *C, const float * __restrict__ A, const float * __restrict__ B,
        const unsigned size)
{
    /********************************************************************
    Determine element index of each thread
    Compute the sum of a pair of elements from the input vectors
    Write the computed sum to the output vector at the correct index
    ********************************************************************/





}
